#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include "drillx.h"
#include "equix.h"
#include "hashx.h"
#include "equix/src/context.h"
#include "equix/src/solver.h"
#include "equix/src/solver_heap.h"
#include "hashx/src/context.h"

// const int BATCH_SIZE = 512 * 1;

extern "C" void hash(uint8_t *challenge, uint8_t *nonce, uint8_t *digest, int batch_size)
{
    // Generate a hash function for each (challenge, nonce)
    hashx_ctx **ctxs;
    if (hipMallocManaged(&ctxs, batch_size * sizeof(hashx_ctx *)) != hipSuccess)
    {
        printf("Failed to allocate managed memory for ctxs\n");
        return;
    }
    uint8_t seed[40];
    memcpy(seed, challenge, 32);
    for (int i = 0; i < batch_size; i++)
    {
        uint64_t nonce_offset = *((uint64_t *)nonce) + i;
        memcpy(seed + 32, &nonce_offset, 8);
        ctxs[i] = hashx_alloc(HASHX_INTERPRETED);
        if (!ctxs[i] || !hashx_make(ctxs[i], seed, 40))
        {
            // printf("Failed to make hash\n");
        }
    }

    // Allocate space to hold on to hash values (~500KB per seed)
    uint64_t **hash_space;
    if (hipMallocManaged(&hash_space, batch_size * sizeof(uint64_t *)) != hipSuccess)
    {
        printf("Failed to allocate managed memory for hash_space\n");
        return;
    }
    for (int i = 0; i < batch_size; i++)
    {
        if (hipMallocManaged(&hash_space[i], INDEX_SPACE * sizeof(uint64_t)) != hipSuccess)
        {
            printf("Failed to allocate managed memory for hash_space[%d]\n", i);
            return;
        }
    }

    // Launch kernel to parallelize hashx operations
    dim3 threadsPerBlock(256);                                                            // 256 threads per block
    dim3 blocksPerGrid((65536 * batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x); // enough blocks to cover batch
    do_hash_stage0i<<<blocksPerGrid, threadsPerBlock>>>(ctxs, hash_space, batch_size);
    hipDeviceSynchronize();

    // equix_ctx
    equix_ctx **eq_ctxs;
    if (hipMallocManaged(&eq_ctxs, batch_size * sizeof(equix_ctx *)) != hipSuccess)
    {
        printf("Failed to allocate managed memory for equix_ctx\n");
    }
    for (int i = 0; i < batch_size; i++)
    {
        eq_ctxs[i] = equix_alloc(EQUIX_CTX_SOLVE);

        if (eq_ctxs[i] == nullptr)
        {
            printf("Failed to allocate equix context\n");
            return;
        }
        else
        {
            eq_ctxs[i]->hash = hash_space[i];
        }
    }
    // digest
    uint8_t *fp_device_digest;
    hipMalloc((float **)&fp_device_digest, batch_size * sizeof(uint8_t) * 16);
    if (fp_device_digest != NULL)
    {
        hipMemset(fp_device_digest, 0, batch_size * sizeof(uint8_t) * 16);
    }

    do_remain_stage<<<batch_size / 128, 128>>>(eq_ctxs, fp_device_digest, batch_size);
    hipDeviceSynchronize();

    // copy to host
    hipMemcpy(digest, fp_device_digest, batch_size * sizeof(uint8_t) * 16, hipMemcpyDeviceToHost);

    // Free memory
    for (int i = 0; i < batch_size; i++)
    {
        hashx_free(ctxs[i]);
        equix_free(eq_ctxs[i]);
        hipFree(hash_space[i]);
    }
    hipFree(hash_space);
    hipFree(fp_device_digest);
    hipFree(ctxs);
    hipFree(eq_ctxs);

    // Generate a hash function for each (challenge, nonce)

    // Print errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        printf("Error at file:%s, line:%d\n", __FILE__, __LINE__);
    }
}

__global__ void do_hash_stage0i(hashx_ctx **ctxs, uint64_t **hash_space, int batch_size)
{
    uint32_t item = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t batch_idx = item / INDEX_SPACE;
    uint32_t i = item % INDEX_SPACE;
    if (batch_idx < batch_size)
    {
        hash_stage0i(ctxs[batch_idx], hash_space[batch_idx], i);
    }
}

__global__ void do_remain_stage(equix_ctx **ctxs, uint8_t *digest, int batch_size)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < batch_size)
    {
        equix_ctx *ctx = ctxs[i];
        equix_solution solutions[EQUIX_MAX_SOLS];
        uint32_t num_sols = equix_solver_solve(ctx->hash, ctx->heap, solutions);

        if (num_sols > 0)
        {
            memcpy(digest + (i * 16), solutions[0].idx, sizeof(solutions[0].idx));
        }
        else
        {
            memset(digest + (i * 16), 0, sizeof(solutions[0].idx));
        }
    }
}
