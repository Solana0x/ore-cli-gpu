#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include "drillx.h"
#include "equix.h"
#include "hashx.h"
#include "equix/src/context.h"
#include "equix/src/solver.h"
#include "equix/src/solver_heap.h"
#include "hashx/src/context.h"

const int BATCH_SIZE = 4096;

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(err); \
        } \
    } while (0)

extern "C" void hash(uint8_t *challenge, uint8_t *nonce, uint64_t *out) {
    hashx_ctx** ctxs;
    uint64_t** hash_space;

    CUDA_CHECK(hipMallocManaged(&ctxs, BATCH_SIZE * sizeof(hashx_ctx*)));
    CUDA_CHECK(hipMallocManaged(&hash_space, BATCH_SIZE * sizeof(uint64_t*)));

    for (int i = 0; i < BATCH_SIZE; i++) {
        CUDA_CHECK(hipMallocManaged(&hash_space[i], INDEX_SPACE * sizeof(uint64_t)));
    }

    uint8_t seed[40];
    memcpy(seed, challenge, 32);

    // Create CUDA stream for asynchronous operations
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    for (int i = 0; i < BATCH_SIZE; i++) {
        uint64_t nonce_offset = *((uint64_t*)nonce) + i;
        memcpy(seed + 32, &nonce_offset, 8);
        ctxs[i] = hashx_alloc(HASHX_INTERPRETED);
        if (!ctxs[i] || !hashx_make(ctxs[i], seed, 40)) {
            for (int j = 0; j <= i; j++) {
                hashx_free(ctxs[j]);
                CUDA_CHECK(hipFree(hash_space[j]));
            }
            CUDA_CHECK(hipFree(ctxs));
            CUDA_CHECK(hipStreamDestroy(stream));
            return;
        }
    }

    dim3 threadsPerBlock(1024);  // Increased for 4090 GPU
    dim3 blocksPerGrid((BATCH_SIZE * INDEX_SPACE + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // Launch kernel with the created stream
    do_hash_stage0i<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(ctxs, hash_space);
    CUDA_CHECK(hipGetLastError()); // Check for launch errors

    // Synchronize the stream instead of the entire device
    CUDA_CHECK(hipStreamSynchronize(stream));

    for (int i = 0; i < BATCH_SIZE; i++) {
        CUDA_CHECK(hipMemcpyAsync(out + i * INDEX_SPACE, hash_space[i], INDEX_SPACE * sizeof(uint64_t), hipMemcpyDeviceToHost, stream));
    }

    CUDA_CHECK(hipStreamSynchronize(stream)); // Ensure all memcpyAsync operations are done

    for (int i = 0; i < BATCH_SIZE; i++) {
        hashx_free(ctxs[i]);
        CUDA_CHECK(hipFree(hash_space[i]));
    }
    CUDA_CHECK(hipFree(ctxs));

    // Destroy the CUDA stream
    CUDA_CHECK(hipStreamDestroy(stream));
}

__global__ void do_hash_stage0i(hashx_ctx** ctxs, uint64_t** hash_space) {
    uint32_t item = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t batch_idx = item / INDEX_SPACE;
    uint32_t i = item % INDEX_SPACE;
    if (batch_idx < BATCH_SIZE) {
        hash_stage0i(ctxs[batch_idx], hash_space[batch_idx], i);
    }
}

extern "C" void solve_all_stages(uint64_t *hashes, uint8_t *out, uint32_t *sols, int num_sets) {
    uint64_t *d_hashes;
    solver_heap *d_heaps;
    equix_solution *d_solutions;
    uint32_t *d_num_sols;

    CUDA_CHECK(hipMallocManaged(&d_hashes, num_sets * INDEX_SPACE * sizeof(uint64_t)));
    CUDA_CHECK(hipMallocManaged(&d_heaps, num_sets * sizeof(solver_heap)));
    CUDA_CHECK(hipMallocManaged(&d_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution)));
    CUDA_CHECK(hipMallocManaged(&d_num_sols, num_sets * sizeof(uint32_t)));

    equix_solution *h_solutions;
    uint32_t *h_num_sols;
    CUDA_CHECK(hipHostAlloc(&h_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&h_num_sols, num_sets * sizeof(uint32_t), hipHostMallocDefault));

    CUDA_CHECK(hipMemcpy(d_hashes, hashes, num_sets * INDEX_SPACE * sizeof(uint64_t), hipMemcpyHostToDevice));

    int threadsPerBlock = 1024;  // Adjusted for 4090 GPU
    int blocksPerGrid = (num_sets + threadsPerBlock - 1) / threadsPerBlock;

    // Create CUDA stream for asynchronous operations
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    solve_all_stages_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_hashes, d_heaps, d_solutions, d_num_sols);
    CUDA_CHECK(hipGetLastError());

    // Synchronize the stream instead of the entire device
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipMemcpyAsync(h_solutions, d_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(h_num_sols, d_num_sols, num_sets * sizeof(uint32_t), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream)); // Ensure all memcpyAsync operations are done

    for (int i = 0; i < num_sets; i++) {
        sols[i] = h_num_sols[i];
        if (h_num_sols[i] > 0) {
            memcpy(out + i * sizeof(equix_solution), &h_solutions[i * EQUIX_MAX_SOLS], sizeof(equix_solution));
        }
    }

    CUDA_CHECK(hipFree(d_hashes));
    CUDA_CHECK(hipFree(d_heaps));
    CUDA_CHECK(hipFree(d_solutions));
    CUDA_CHECK(hipFree(d_num_sols));

    CUDA_CHECK(hipHostFree(h_solutions));
    CUDA_CHECK(hipHostFree(h_num_sols));

    // Destroy the CUDA stream
    CUDA_CHECK(hipStreamDestroy(stream));
}
